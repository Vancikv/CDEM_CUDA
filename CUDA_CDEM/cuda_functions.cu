#include "hip/hip_runtime.h"
#include "cuda_functions.cuh"
#include "aux_functions.h"
#include <fstream>
#include <istream>

hipError_t element_step_with_CUDA(FLOAT_TYPE * u, FLOAT_TYPE * v, FLOAT_TYPE * a,
	FLOAT_TYPE * load, FLOAT_TYPE * supports, int * neighbors, FLOAT_TYPE * n_vects, FLOAT_TYPE * K, FLOAT_TYPE * C, FLOAT_TYPE * Mi,
	FLOAT_TYPE * Kc, int n_els, int n_nds, int n_nodedofs, int stiffdim,
	FLOAT_TYPE t_load, FLOAT_TYPE t_max, int maxiter, char * outfile, int output_frequency, int gridDim, int blockDim)
{
	// Declare device vars
	FLOAT_TYPE * dev_u, *dev_v, *dev_a, *dev_load, *dev_supports, *dev_n_vects, *dev_K, *dev_C, *dev_Mi,
		*dev_Kc;
	FLOAT_TYPE * dev_u_last, *dev_v_last;
	int * dev_neighbors;
	FLOAT_TYPE dt = t_max / maxiter;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);


	hipError_t cudaStatus;
	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}
	dev_u = copy2gpu(u, 2 * n_nds);
	dev_v = copy2gpu(v, 2 * n_nds);
	dev_a = copy2gpu(a, 2 * n_nds);
	dev_u_last = copy2gpu(u, 2 * n_nds);
	dev_v_last = copy2gpu(v, 2 * n_nds);
	dev_load = copy2gpu(load, 2 * n_nds);
	dev_supports = copy2gpu(supports, 2 * n_nds);
	dev_n_vects = copy2gpu(n_vects, 4 * n_nds);
	dev_neighbors = copy2gpu(neighbors, 2 * n_nds);
	dev_K = copy2gpu(K, stiffdim * stiffdim * n_els);
	dev_C = copy2gpu(C, stiffdim * n_els);
	dev_Mi = copy2gpu(Mi, stiffdim * n_els);
	dev_Kc = copy2gpu(Kc, 4);

	int nblocks = gridDim;
	if (gridDim*blockDim <= 0)
	{
		nblocks = ((n_nds*n_nodedofs) / blockDim) + 1;
	}
	dim3 dimBlock(blockDim, 1, 1);
	dim3 dimGrid(nblocks,1,1);
	std::cout << "Running kernels in " << nblocks << " blocks of " << blockDim << "threads each." << std::endl;

	int i, j;
	for (i = 1; i <= maxiter; i++)
	{
		memorize_and_increment << <dimGrid, dimBlock >> >(dev_u, dev_v, dev_a, dev_u_last, dev_v_last, n_nodedofs*n_nds, dt);

		if (i > 1) // Relaxation step
		{
			// hipDeviceSynchronize waits for the kernel to finish, and returns
			// any errors encountered during the launch.
			CUDA_SYNCHRO(cudaStatus)

			element_step_kernel << < dimGrid, dimBlock >> > (dev_u, dev_v, dev_a, dev_load, dev_supports, dev_neighbors,
				dev_n_vects, dev_K, dev_C, dev_Mi, dev_Kc, n_els, n_nds, n_nodedofs, stiffdim, load_function(dt*i / t_load));

			CUDA_ERRORCHCK(cudaStatus)
			CUDA_SYNCHRO(cudaStatus)

			increment << <dimGrid, dimBlock >> >(dev_u, dev_v, dev_a, dev_u_last, dev_v_last, n_nodedofs*n_nds, dt);
		}
		CUDA_SYNCHRO(cudaStatus)

		element_step_kernel << < dimGrid, dimBlock >> > (dev_u, dev_v, dev_a, dev_load, dev_supports, dev_neighbors,
		dev_n_vects, dev_K, dev_C, dev_Mi, dev_Kc, n_els, n_nds, n_nodedofs, stiffdim, load_function(dt*i/t_load));

		CUDA_ERRORCHCK(cudaStatus)
		CUDA_SYNCHRO(cudaStatus)

		if (((i%output_frequency) == 0) || (i == 1))
		{
			// Copy output from GPU buffer to host memory.
			cudaStatus = hipMemcpy(u, dev_u, 2 * n_nds * sizeof(FLOAT_TYPE), hipMemcpyDeviceToHost);
			if (cudaStatus != hipSuccess) {
				fprintf(stderr, "hipMemcpy failed!");
				goto Error;
			}
			// Copy output from GPU buffer to host memory.
			cudaStatus = hipMemcpy(v, dev_v, 2 * n_nds * sizeof(FLOAT_TYPE), hipMemcpyDeviceToHost);
			if (cudaStatus != hipSuccess) {
				fprintf(stderr, "hipMemcpy failed!");
				goto Error;
			}
			// Copy output from GPU buffer to host memory.
			cudaStatus = hipMemcpy(a, dev_a, 2 * n_nds * sizeof(FLOAT_TYPE), hipMemcpyDeviceToHost);
			if (cudaStatus != hipSuccess) {
				fprintf(stderr, "hipMemcpy failed!");
				goto Error;
			}
			// hipDeviceSynchronize waits for the kernel to finish, and returns
			// any errors encountered during the launch.
			cudaStatus = hipDeviceSynchronize();
			if (cudaStatus != hipSuccess) {
				fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching element_step_kernel!\n", cudaStatus);
				goto Error;
			}
			char fn[100];
			sprintf(fn,"%s%d.txt",outfile, i);
			std::ofstream f(fn);

			f << n_nds << " " << n_els << " " << i*dt << " " << load_function(dt*i / t_load) << std::endl;
			for (j = 0; j < n_nds; j++)
			{
				f << "node " << (j + 1) << " x y " << u[2 * j] << " " << u[2*j+1]
					<< " " << v[2 * j] << " " << v[2 * j + 1] << " " << a[2 * j] << " " << a[2 * j + 1] << std::endl;
			}
		}
	}
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	float elapsedTime;
	hipEventElapsedTime(&elapsedTime, start, stop);
	printf("GPU solve time %3.5f[s]\n", elapsedTime / 1000);
	hipEventDestroy(start);
	hipEventDestroy(stop);
	return cudaStatus;

Error:
	hipFree(dev_u);
	hipFree(dev_v);
	hipFree(dev_a);
	hipFree(dev_supports);
	hipFree(dev_neighbors);
	hipFree(dev_n_vects);
	hipFree(dev_K);
	hipFree(dev_Mi);
	hipFree(dev_C);
	hipFree(dev_Kc);

	return cudaStatus;
}

__global__ void element_step_kernel(FLOAT_TYPE * u, FLOAT_TYPE * v, FLOAT_TYPE * a, FLOAT_TYPE * load, FLOAT_TYPE * supports, int * neighbors, 
	FLOAT_TYPE * n_vects, FLOAT_TYPE * K, FLOAT_TYPE * C, FLOAT_TYPE * Mi, FLOAT_TYPE * Kc, int n_els, int n_nds, int n_nodedofs, int stiffdim, 
	FLOAT_TYPE loadfunc)
{
	int dofid = threadIdx.x + blockIdx.x * blockDim.x; // thread id - global number of dof

	while (dofid<n_nds*n_nodedofs)
	{
		int eid = dofid / stiffdim; // global number of element
		int nid = (dofid / n_nodedofs) * n_nodedofs; // number of dof 1 of this node
		int ned = dofid % stiffdim; // number of dof within element
		int mdim = stiffdim*stiffdim; // number of elements of the stiffness matrix
		int i;
		FLOAT_TYPE kc11 = Kc[0];
		FLOAT_TYPE kc21 = Kc[1];
		FLOAT_TYPE kc12 = Kc[2];
		FLOAT_TYPE kc22 = Kc[3];

		// Element stiffness force:
		FLOAT_TYPE F_k_e = 0;
		for (i = 0; i < stiffdim; i++)
		{
			F_k_e += -K[eid*mdim + i*stiffdim + ned] * u[eid*stiffdim + i];
		}
		// Contact stiffness force:
		FLOAT_TYPE F_k_c = 0;
		for (i = 0; i < 2; i++)
		{
			int nbr = neighbors[nid + i];
			if (nbr != 0)
			{
				FLOAT_TYPE t11 = n_vects[4 * (dofid / n_nodedofs) + 2 * i];
				FLOAT_TYPE t12 = n_vects[4 * (dofid / n_nodedofs) + 2 * i + 1];
				FLOAT_TYPE t21 = -t12;
				FLOAT_TYPE t22 = t11;
				FLOAT_TYPE du_x = u[(nbr - 1)*n_nodedofs] - u[nid];
				FLOAT_TYPE du_y= u[(nbr - 1)*n_nodedofs+1] - u[nid+1];
				if (dofid == nid) // X-component
				{
					F_k_c += du_x * (t11*(t11*kc11 + t21*kc21) + t21*(t11*kc12 + t21*kc22)) + du_y * (t12*(t11*kc11 + t21*kc21) + t22*(t11*kc12 + t21*kc22)); // T_T * Kc * T * du_g
				}
				else // Y-component
				{
					F_k_c += du_x * (t11*(t12*kc11 + t22*kc21) + t21*(t12*kc12 + t22*kc22)) + du_y * (t12*(t12*kc11 + t22*kc21) + t22*(t12*kc12 + t22*kc22)); // T_T * Kc * T * du_g
				}
			}
		}
		// Damping force:
		FLOAT_TYPE F_c = -C[dofid] * v[dofid];
		// Reaction force
		FLOAT_TYPE F_r = supports[dofid] * (-F_k_e - F_k_c - F_c - loadfunc*load[dofid]);
		a[dofid] = Mi[dofid] * (F_k_e + F_k_c + F_r + F_c + loadfunc*load[dofid]);
		dofid += gridDim.x * blockDim.x;
	}
}

__global__ void memorize_and_increment(FLOAT_TYPE * u, FLOAT_TYPE * v, FLOAT_TYPE * a, FLOAT_TYPE * u_last, FLOAT_TYPE * v_last, int vdim, FLOAT_TYPE dt)
{
	int dofid = threadIdx.x + blockIdx.x * blockDim.x; // thread id - global number of dof
	while (dofid<vdim)
	{
		u_last[dofid] = u[dofid];
		u[dofid] += dt*v[dofid] + 0.5*dt*dt*a[dofid];
		v_last[dofid] = v[dofid];
		v[dofid] += dt*a[dofid];
		dofid += gridDim.x * blockDim.x;
	}
}

__global__ void increment(FLOAT_TYPE * u, FLOAT_TYPE * v, FLOAT_TYPE * a, FLOAT_TYPE * u_last, FLOAT_TYPE * v_last, int vdim, FLOAT_TYPE dt)
{
	int dofid = threadIdx.x + blockIdx.x * blockDim.x; // thread id - global number of dof
	while (dofid < vdim)
	{
		u[dofid] = u_last[dofid] + dt*v_last[dofid] + 0.5*dt*dt*a[dofid];
		v[dofid] = v_last[dofid] + dt*a[dofid];
		dofid += gridDim.x * blockDim.x;
	}
}