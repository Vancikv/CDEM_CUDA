#include "hip/hip_runtime.h"
#include "cuda_functions.cuh"
#include "aux_functions.h"
#include <fstream>
#include <istream>

hipError_t element_step_with_CUDA(double * u, double * v, double * a,
	double * load, double * supports, int * neighbors, double * n_vects, double * K, double * C, double * Mi,
	double * Kc, int n_els, int n_nds, int n_nodedofs, int stiffdim, 
	double t_load, double t_max, int maxiter,char * outfile, int output_frequency)
{
	// Declare device vars
	double * dev_u, *dev_v, *dev_a, *dev_load, *dev_supports, * dev_n_vects, *dev_K, *dev_C, *dev_Mi,
	* dev_Kc;
	int * dev_neighbors;
	float dt = t_max / maxiter;

	hipError_t cudaStatus;
	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}
	// Allocate GPU buffers.
	cudaStatus = hipMalloc((void**)&dev_u, 2*n_nds * sizeof(double));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}
	cudaStatus = hipMalloc((void**)&dev_v, 2 * n_nds * sizeof(double));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}
	cudaStatus = hipMalloc((void**)&dev_a, 2 * n_nds * sizeof(double));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}
	cudaStatus = hipMalloc((void**)&dev_load, 2 * n_nds * sizeof(double));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}
	cudaStatus = hipMalloc((void**)&dev_supports, 2 * n_nds * sizeof(double));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}
	cudaStatus = hipMalloc((void**)&dev_n_vects, 4 * n_nds * sizeof(double));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}
	cudaStatus = hipMalloc((void**)&dev_neighbors, 2 * n_nds * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}
	cudaStatus = hipMalloc((void**)&dev_K, 64 * n_els * sizeof(double));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}
	cudaStatus = hipMalloc((void**)&dev_C, 8 * n_els * sizeof(double));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}
	cudaStatus = hipMalloc((void**)&dev_Mi, 8 * n_els * sizeof(double));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}
	cudaStatus = hipMalloc((void**)&dev_Kc, 4 * sizeof(double));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_supports, supports, 2 * n_nds * sizeof(double), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}
	cudaStatus = hipMemcpy(dev_neighbors, neighbors, 2 * n_nds * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}
	cudaStatus = hipMemcpy(dev_n_vects, n_vects, 4* n_nds * sizeof(double), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}
	cudaStatus = hipMemcpy(dev_K, K, 64 * n_els * sizeof(double), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}
	cudaStatus = hipMemcpy(dev_Mi, Mi, 8 * n_els * sizeof(double), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}
	cudaStatus = hipMemcpy(dev_C, C, 8 * n_els * sizeof(double), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}
	cudaStatus = hipMemcpy(dev_Kc, Kc, 4 * sizeof(double), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	int warps_per_block = 4;
	int threads_per_block = 32*warps_per_block;
	int nblocks = ((n_nds*n_nodedofs) / threads_per_block) + 1;
	dim3 dimBlock(threads_per_block);
	dim3 dimGrid(nblocks);

	int i, j;
	for (i = 0; i < maxiter; i++)
	{
		for (j = 0; j < n_nds*n_nodedofs; j++)
		{
			u[j] += dt*v[j] + 0.5*dt*dt*a[j];
			v[j] += dt*a[j];
		}
		cudaStatus = hipMemcpy(dev_u, u, 2 * n_nds * sizeof(double), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
			goto Error;
		}
		cudaStatus = hipMemcpy(dev_v, v, 2 * n_nds * sizeof(double), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
			goto Error;
		}
		cudaStatus = hipMemcpy(dev_a, a, 2 * n_nds * sizeof(double), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
			goto Error;
		}

		element_step_kernel <<< dimGrid, dimBlock >>> (dev_u, dev_v, dev_a, dev_load, dev_supports, dev_neighbors,
		dev_n_vects, dev_K, dev_C, dev_Mi, dev_Kc, n_els, n_nds, n_nodedofs, stiffdim, load_function(dt*i/t_load));

		// Check for any errors launching the kernel
		cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "element_step_kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
			goto Error;
		}

		// hipDeviceSynchronize waits for the kernel to finish, and returns
		// any errors encountered during the launch.
		cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching element_step_kernel!\n", cudaStatus);
			goto Error;
		}

		// Copy output from GPU buffer to host memory.
		cudaStatus = hipMemcpy(u, dev_u, 2 * n_nds * sizeof(double), hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
			goto Error;
		}
		cudaStatus = hipMemcpy(v, dev_v, 2 * n_nds * sizeof(double), hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
			goto Error;
		}
		cudaStatus = hipMemcpy(a, dev_a, 2 * n_nds * sizeof(double), hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
			goto Error;
		}

		if ((i%output_frequency) == 0)
		{
			char fn[100];
			sprintf(fn,"%s%d.txt",outfile, i);
			std::ofstream f(fn);

			f << n_nds << " " << n_els << " " << i*dt << std::endl;
			for (j = 0; j < n_nds; j++)
			{
				f << "node " << (i + 1) << " x y " << u[2 * j] << " " << u[2*j+1]
					<< " " << v[2 * j] << " " << v[2 * j + 1] << " " << a[2 * j] << " " << a[2 * j + 1] << std::endl;
			}
		}
	}
Error:
	hipFree(dev_u);
	hipFree(dev_v);
	hipFree(dev_a);
	hipFree(dev_supports);
	hipFree(dev_neighbors);
	hipFree(dev_n_vects);
	hipFree(dev_K);
	hipFree(dev_Mi);
	hipFree(dev_C);
	hipFree(dev_Kc);

	return cudaStatus;
}

__global__ void element_step_kernel(double * u, double * v, double * a, double * load, double * supports, int * neighbors, 
	double * n_vects, double * K, double * C, double * Mi, double * Kc, int n_els, int n_nds, int n_nodedofs, int stiffdim, 
	float loadfunc)
{
	int tid = threadIdx.x + blockIdx.x * blockDim.x; // thread id - global number of dof
	if (tid < n_nds*n_nodedofs)
	{
		int eid = tid / stiffdim; // global number of element
		int nid = (tid / n_nodedofs) * n_nodedofs; // number of dof 1 of this node
		int ned = tid % stiffdim; // number of dof within element
		int mdim = stiffdim*stiffdim; // number of elements of the stiffness matrix
		int i;
		double kc11 = Kc[0];
		double kc21 = Kc[1];
		double kc12 = Kc[2];
		double kc22 = Kc[3];

		// Element stiffness force:
		double F_k_e = 0;
		for (i = 0; i < stiffdim; i++)
		{
			F_k_e += -K[eid*mdim + i*stiffdim + ned] * u[eid*stiffdim + i];
		}
		// Contact stiffness force:
		double F_k_c = 0;
		for (i = 0; i < 2; i++)
		{
			int nbr = neighbors[nid + i];
			if (nbr != 0)
			{
				double t11 = n_vects[4*(tid/n_nodedofs)+2*i];
				double t12 = n_vects[4 * (tid / n_nodedofs) + 2 * i+1];
				double t21 = -t12;
				double t22 = t11;
				double du_x = u[(nbr - 1)*n_nodedofs] - u[nid];
				double du_y= u[(nbr - 1)*n_nodedofs+1] - u[nid+1];
				if (tid == nid) // X-component
				{
					F_k_c += du_x * (t11*(t11*kc11 + t21*kc21) + t21*(t11*kc12 + t21*kc22)) + du_y * (t12*(t11*kc11 + t21*kc21) + t22*(t11*kc12 + t21*kc22)); // T_T * Kc * T * du_g
				}
				else // Y-component
				{
					F_k_c += du_x * (t11*(t12*kc11 + t22*kc21) + t21*(t12*kc12 + t22*kc22)) + du_y * (t12*(t12*kc11 + t22*kc21) + t22*(t12*kc12 + t22*kc22)); // T_T * Kc * T * du_g
				}
			}
		}
		// Damping force:
		double F_c = -C[tid] * v[tid];
		// Reaction force
		double F_r = supports[tid] * (-F_k_e - F_k_c - F_c - loadfunc*load[tid]);
		a[tid] = Mi[tid] * (F_k_e + F_k_c + F_r + F_c + loadfunc*load[tid]);
	}
}